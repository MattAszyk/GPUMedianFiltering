
#include <hip/hip_runtime.h>
/*
Our source have format ARGB per pixel, so we can test two approches here.
1. Kernel calculates all values per pixel
2. Kernel calculates one value (A/R/G/B) separately
*/

extern "C" __device__
void
sort(unsigned char src[], int size = 9)
{
    // as long we have only 9 elements there is no reason to implement more efficient sorting algorithm than bubble sort.
    unsigned char temp;
    for (int i = 0; i < size - 1; ++i)
    {
        for(int j = 0; j < size - i - 1; ++j)
        {
            if(src[j] > src[j+1])
            {
                temp = src[j];
                src[j] = src[j+1];
                src[j+1] = temp;
            }
        }
    }
}


// We want to have one kernel per pixel. Not for per color. So we need move every i * 4;
extern "C" __global__ void
CalculatePerPixel(const unsigned char *source, unsigned char *destination, int totalSize, int height, int width, int stride)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    i *= 4;
    if(i > totalSize) return;
    //Our (X,Y) 
    int h = i / stride;
    int w = (i - h * stride) / 4;
    if(h+1 >= height || h - 1 < 0) return;
    if(w+1 >= width || w - 1 < 0) return;
    
    unsigned char A[9];
    unsigned char R[9];
    unsigned char G[9];
    unsigned char B[9];
    
    //GetKernel
    //byte B, byte G, byte R, byte A
    int pos = 0;
    int index = 0;
    for(int y = h - 1; y <= h + 1; ++y)
    {
        for(int x = w - 1; x <= w + 1; ++x)
        {
            pos = y * stride + 4 * x;
            B[index] = source[pos];
            G[index] = source[pos+1];
            R[index] = source[pos+2];
            A[index] = source[pos+3];
            index++;
        }
    }
    
    sort(A);
    sort(R);
    sort(B);
    sort(G);
    
    *(destination + i++) = B[4];
    *(destination + i++) = G[4];
    *(destination + i++) = R[4];
    *(destination + i++) = A[4];
}
